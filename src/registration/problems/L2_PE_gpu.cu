#include "hip/hip_runtime.h"
// STD Includes
#include <chrono>
#include <iostream>
#include <numeric>

// PCL Includes
#include <pcl/gpu/containers/device_array.h>

// 添加cuda库
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

// CUDA Helper Includes
#include "hip/hip_runtime_api.h"

// Registration Includes
#include "L2_PE_gpu.cuh"

__global__ void calculateOnePointPotentialEnergy(pcl::gpu::PtrSz<pcl::PointXYZ> input,
                                                 pcl::gpu::PtrSz<pcl::PointXYZ> target,
                                                 thrust::device_ptr<float> sums, int input_size,
                                                 int target_size)
{
  float localSum = 0;
  int inputIndex = blockDim.x * blockIdx.x + threadIdx.x;
  if (inputIndex >= input_size) {
    return;
  }
  for (int i = 0; i < target_size; i++) {
    float dx = __powf(std::abs(input[inputIndex].x - target[i].x), 2);
    float dy = __powf(std::abs(input[inputIndex].y - target[i].y), 2);
    float dz = __powf(std::abs(input[inputIndex].z - target[i].z), 2);
    float dis = dx + dy + dz;
//    printf("%f \n", dx);
//    printf("%f \n", dy);
//    printf("%f \n", dz);
    if (dis < 0.0000001f) {
      dis = 0.0000001f;
    }
    localSum += -1.0f / dis;
  }
  sums[inputIndex] = localSum;
}

float cloud2GPU(pcl::gpu::DeviceArray<pcl::PointXYZ>& input,
                pcl::gpu::DeviceArray<pcl::PointXYZ>& target, float& fitnessScore,
                std::vector<int>& inputIndex, std::vector<int>& targetIndex)
{
  int n = input.size();
  thrust::host_vector<float> cpuSums(inputIndex.size());
  thrust::device_vector<float> gpuSums = cpuSums;

  int thread = 64;
  int block = (n + thread + 1) / thread;
  auto begin = std::chrono::high_resolution_clock::now();

  calculateOnePointPotentialEnergy<<<block, thread>>>(input, target, gpuSums.data(),
                                                      inputIndex.size(), targetIndex.size());

  cpuSums = gpuSums;
  fitnessScore = std::accumulate(cpuSums.cbegin(), cpuSums.cend(), 0.0f);
  checkCudaErrors(hipDeviceSynchronize());
  return false;
}
