#include "hip/hip_runtime.h"
#include "foo.cuh"
#include <iostream>
// 添加cuda库
#include "../registration/hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include ""

// 定义核函数 __global__为声明关键字
template <typename T>
__global__ void matAdd_cuda(T* a, T* b, T* sum)
{
  // blockIdx代表block的索引,blockDim代表block的大小，threadIdx代表thread线程的索引，因此对于一维的block和thread索引的计算方式如下
  static int v = 0;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  printf("%f",sum[i]);
  sum[i] = a[i] * a[i] + b[i] * b[i] + a[i] * b[i];
  sum[i] /= sum[i];
  sum[i] = 1000.0f;
  v++;
}

__global__ void kernel()
{
  printf("Hello, world!\n");
}

// 核函数用模板不会报错，模板名字是具有链接的，但它们不能具有C链接，因此不能用在供调用的函数上
float* matAdd(float* a, float* b, int length)
{
  int device = 0;        // 设置使用第0块GPU进行运算
  hipSetDevice(device); // 设置运算显卡
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, device); // 获取对应设备属性
  kernel<<<1, 1>>>();
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
  int threadMaxSize = devProp.maxThreadsPerBlock; // 每个线程块的最大线程数
  int blockSize = (length + threadMaxSize - 1) /
                  threadMaxSize; // 计算Block大小,block一维度是最大的，一般不会溢出
  dim3 thread(threadMaxSize);    // 设置thread
  dim3 block(blockSize);         // 设置block
  int size = length * sizeof(float); // 计算空间大小
  float* sum = (float*)malloc(size); // 开辟动态内存空间
  // 开辟显存空间
  float *sumGPU, *aGPU, *bGPU;
  hipMalloc((void**)&sumGPU, size);
  hipMalloc((void**)&aGPU, size);
  hipMalloc((void**)&bGPU, size);
  // 内存->显存
  hipMemcpy((void*)aGPU, (void*)a, size, hipMemcpyHostToDevice);
  hipMemcpy((void*)bGPU, (void*)b, size, hipMemcpyHostToDevice);
  // 运算
  for (int i = 0; i < length; i++) {
    matAdd_cuda<float><<<block, thread>>>(aGPU, bGPU, sumGPU);
  }
  //  std::cout<<v<<std::endl;
  // hipDeviceSynchronize();
  // 显存->内存
  hipMemcpy(sum, sumGPU, size, hipMemcpyDeviceToHost);
  printf("%f", sum[12]);
  hipDeviceSynchronize();
  // 释放显存
  hipFree(sumGPU);
  hipFree(aGPU);
  hipFree(bGPU);
  return sum;
}
